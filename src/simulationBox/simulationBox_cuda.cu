/*****************************************************************************
<GPL_HEADER>

    PQ
    Copyright (C) 2023-now  Jakob Gamper

    This program is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.

    This program is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.

    You should have received a copy of the GNU General Public License
    along with this program.  If not, see <http://www.gnu.org/licenses/>.

<GPL_HEADER>
******************************************************************************/

#include "hip/hip_runtime.h"
#include "simulationBox_cuda.cuh"
#include "simulationBox.hpp"

using namespace simulationBox;

/**
 * @brief Constructor
 * 
 * @param numAtoms
 */
SimulationBoxCuda::SimulationBoxCuda(size_t numAtoms){
    // set number of atoms
    this->numAtoms = numAtoms;
    // allocate memory on device
    hipMalloc((void **)&_atomTypes, numAtoms * sizeof(size_t));
    hipMalloc((void **)&_molTypes, numAtoms * sizeof(size_t));
    hipMalloc((void **)&_moleculeIndices, numAtoms * sizeof(size_t));
    hipMalloc((void **)&_internatGlobalVDWTypes, numAtoms * sizeof(size_t));
    hipMalloc((void **)&_positions, numAtoms * 3 * sizeof(double));
    hipMalloc((void **)&_velocities, numAtoms * 3 * sizeof(double));
    hipMalloc((void **)&_forces, numAtoms * 3 * sizeof(double));
    hipMalloc((void **)&_shiftForeces, numAtoms * 3 * sizeof(double));
    hipMalloc((void **)&_pratialCharges, numAtoms * sizeof(double));
    hipMalloc((void **)&_masses, numAtoms * sizeof(double));
    hipMalloc((void **)&_boxDimensions, 3 * sizeof(double));
}

/**
 * @brief Destructor
 */
SimulationBoxCuda::~SimulationBoxCuda(){
    // free memory on device
    hipFree(_atomTypes);
    hipFree(_molTypes);
    hipFree(_moleculeIndices);
    hipFree(_internatGlobalVDWTypes);
    hipFree(_positions);
    hipFree(_velocities);
    hipFree(_forces);
    hipFree(_shiftForeces);
    hipFree(_pratialCharges);
    hipFree(_masses);
    hipFree(_boxDimensions);
}

/**
 * @brief Transfer data to device
 */
void SimulationBoxCuda::transferDataToDevice(SimulationBox &simulationBox){
    // transfer data to device
    transferAtomTypesFromSimulationBox(simulationBox);
    transferMolTypesFromSimulationBox(simulationBox);
    transferMoleculeIndicesFromSimulationBox(simulationBox);
    transferInternalGlobalVDWTypesFromSimulationBox(simulationBox);
    transferPositionsFromSimulationBox(simulationBox);
    transferVelocitiesFromSimulationBox(simulationBox);
    transferForcesFromSimulationBox(simulationBox);
    transferPartialChargesFromSimulationBox(simulationBox);
    transferMassesFromSimulationBox(simulationBox);
    transferBoxDimensionsFromSimulationBox(simulationBox);
}

/**
 * @brief Transfer atom types from simulation box
 */
void SimulationBoxCuda::transferAtomTypesFromSimulationBox(SimulationBox &simulationBox){
    // transfer atom types from simulation box
    hipMemcpy(_atomTypes, simulationBox.getAtomTypes(), numAtoms * sizeof(size_t), hipMemcpyHostToDevice);
}

/**
 * @brief Transfer mol types from simulation box
 */
void SimulationBoxCuda::transferMolTypesFromSimulationBox(SimulationBox &simulationBox){
    // transfer mol types from simulation box
    hipMemcpy(_molTypes, simulationBox.getMolTypes(), numAtoms * sizeof(size_t), hipMemcpyHostToDevice);
}

/**
 * @brief Transfer molecule indices from simulation box
 */
void SimulationBoxCuda::transferMoleculeIndicesFromSimulationBox(SimulationBox &simulationBox){
    // transfer molecule indices from simulation box
    hipMemcpy(_moleculeIndices, simulationBox.getMoleculeIndices(), numAtoms * sizeof(size_t), hipMemcpyHostToDevice);
}

/**
 * @brief Transfer internal global VDW types from simulation box
 */
void SimulationBoxCuda::transferInternalGlobalVDWTypesFromSimulationBox(SimulationBox &simulationBox){
    // transfer internal global VDW types from simulation box
    hipMemcpy(_internatGlobalVDWTypes, simulationBox.getInternalGlobalVDWTypes(), numAtoms * sizeof(size_t), hipMemcpyHostToDevice);
}

/**
 * @brief Transfer positions from simulation box
 */
void SimulationBoxCuda::transferPositionsFromSimulationBox(SimulationBox &simulationBox){
    // transfer positions from simulation box
    hipMemcpy(_positions, simulationBox.getPositions(), numAtoms * 3 * sizeof(double), hipMemcpyHostToDevice);
}

/**
 * @brief Transfer velocities from simulation box
 */
void SimulationBoxCuda::transferVelocitiesFromSimulationBox(SimulationBox &simulationBox){
    // transfer velocities from simulation box
    hipMemcpy(_velocities, simulationBox.getVelocities(), numAtoms * 3 * sizeof(double), hipMemcpyHostToDevice);
}

/**
 * @brief Transfer forces from simulation box
 */
void SimulationBoxCuda::transferForcesFromSimulationBox(SimulationBox &simulationBox){
    // transfer forces from simulation box
    hipMemcpy(_forces, simulationBox.getForces(), numAtoms * 3 * sizeof(double), hipMemcpyHostToDevice);
}

/**
 * @brief Transfer partial charges from simulation box
 */
void SimulationBoxCuda::transferPartialChargesFromSimulationBox(SimulationBox &simulationBox){
    // transfer partial charges from simulation box
    hipMemcpy(_pratialCharges, simulationBox.getPartialCharges(), numAtoms * sizeof(double), hipMemcpyHostToDevice);
}

/**
 * @brief Transfer masses from simulation box
 */
void SimulationBoxCuda::transferMassesFromSimulationBox(SimulationBox &simulationBox){
    // transfer masses from simulation box
    hipMemcpy(_masses, simulationBox.getMasses(), numAtoms * sizeof(double), hipMemcpyHostToDevice);
}

/**
 * @brief Transfer box dimensions from simulation box
 */
void SimulationBoxCuda::transferBoxDimensionsFromSimulationBox(SimulationBox &simulationBox){
    // transfer box dimensions from simulation box
    hipMemcpy(_boxDimensions, simulationBox.getBoxDimensions(), 3 * sizeof(double), hipMemcpyHostToDevice);
}

/**
 * @brief Transfer positions to simulation box
 */
void SimulationBoxCuda::transferPositionsToSimulationBox(SimulationBox &simulationBox){
    // transfer positions to simulation box
    hipMemcpy(simulationBox.getPositions(), _positions, numAtoms * 3 * sizeof(double), hipMemcpyDeviceToHost);
}

/**
 * @brief Transfer velocities to simulation box
 */
void SimulationBoxCuda::transferVelocitiesToSimulationBox(SimulationBox &simulationBox){
    // transfer velocities to simulation box
    hipMemcpy(simulationBox.getVelocities(), _velocities, numAtoms * 3 * sizeof(double), hipMemcpyDeviceToHost);
}

/**
 * @brief Transfer forces to simulation box
 */
void SimulationBoxCuda::transferForcesToSimulationBox(SimulationBox &simulationBox){
    // transfer forces to simulation box
    hipMemcpy(simulationBox.getForces(), _forces, numAtoms * 3 * sizeof(double), hipMemcpyDeviceToHost);
}

/**
 * @brief Transfer shift forces to simulation box
 */
void SimulationBoxCuda::transferShiftForcesToSimulationBox(SimulationBox &simulationBox){
    // transfer shift forces to simulation box
    hipMemcpy(simulationBox.getShiftForces(), _shiftForeces, numAtoms * 3 * sizeof(double), hipMemcpyDeviceToHost);
}

/**
 * @brief Transfer data from device
 */
void SimulationBoxCuda::transferDataFromDevice(SimulationBox &simulationBox){
    // transfer data from device
    transferForcesFromDevice(simulationBox);
    transferShiftForcesFromDevice(simulationBox);
}

/**
 * @brief Transfer forces from device
 */
void SimulationBoxCuda::transferForcesFromDevice(SimulationBox &simulationBox){
    // transfer forces from device
    hipMemcpy(_forces, simulationBox.getForces(), numAtoms * 3 * sizeof(double), hipMemcpyDeviceToHost);
}

/**
 * @brief Transfer shift forces from device
 */
void SimulationBoxCuda::transferShiftForcesFromDevice(SimulationBox &simulationBox){
    // transfer shift forces from device
    hipMemcpy(_shiftForeces, simulationBox.getShiftForces(), numAtoms * 3 * sizeof(double), hipMemcpyDeviceToHost);
}

/**
 * @brief Get struct of simulation box cuda
 */
SimulationBoxCuda_t *SimulationBoxCuda::getSimulationBoxCuda(){
    // create simulation box cuda
    SimulationBoxCuda_t *simulationBoxCuda;

    // set simulation box cuda
    simulationBoxCuda->numAtoms = _numAtoms;
    simulationBoxCuda->numInternalGlobalVDWTypes = _numInternalGlobalVDWTypes;
    simulationBoxCuda->atomTypes = _atomTypes;
    simulationBoxCuda->molTypes = _molTypes;
    simulationBoxCuda->moleculeIndices = _moleculeIndices;
    simulationBoxCuda->internalGlobalVDWTypes = _internatGlobalVDWTypes;
    simulationBoxCuda->positions = _positions;
    simulationBoxCuda->velocities = _velocities;
    simulationBoxCuda->forces = _forces;
    simulationBoxCuda->shiftForeces = _shiftForeces;
    simulationBoxCuda->pratialCharges = _pratialCharges;
    simulationBoxCuda->masses = _masses;
    simulationBoxCuda->boxDimensions = _boxDimensions;

    // return simulation box cuda
    return simulationBoxCuda;
}