#include "hip/hip_runtime.h"
/*****************************************************************************
<GPL_HEADER>

    PQ
    Copyright (C) 2023-now  Jakob Gamper

    This program is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.

    This program is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.

    You should have received a copy of the GNU General Public License
    along with this program.  If not, see <http://www.gnu.org/licenses/>.

<GPL_HEADER>
******************************************************************************/
#include <cstddef>   // for size_t

#include "box.hpp"                // for Box
#include "coulombPotential.hpp"   // for CoulombPotential
#include "hip/hip_runtime.h"
#include "molecule.hpp"              // for Molecule
#include "nonCoulombPair.hpp"        // for NonCoulombPair
#include "nonCoulombPotential.hpp"   // for NonCoulombPotential
#include "physicalData.hpp"          // for PhysicalData
#include "potential.hpp"
#include "simulationBox.hpp"   // for SimulationBox

#include "kernels.cu" 

namespace simulationBox
{
    class CellList;
}   // namespace simulationBox

using namespace potential;

PotentialCuda::~PotentialCuda() = default;

/**
 * @brief Cuda kernel to calculate forces, coulombic and non-coulombic energy
 *
 * @param atomTypes
 * @param moleculeIndices
 * @param internalGlobalVDWTypes
 * @param molTypes
 * @param partialCharges
 * @param positions
 * @param forces
 * @param numberOfMolecules
 * @param coulombEnergy
 * @param nonCoulombEnergy
 */
 __global__ void calculateForcesKernel(
    double3 boxDimensions,
    size_t *atomTypes,
    size_t *moleculeIndices,
    size_t *internalGlobalVDWTypes,
    size_t *molTypes,
    double *partialCharges,
    double *positions,
    double *forces,
    size_t  numberOfMolecules,
    double *coulombEnergy,
    double *nonCoulombEnergy
)
{
    // get thread id
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // check if thread id is smaller than number of molecules
    if (i < numberOfMolecules)
    {
        // forces
        double3 forces_i = {0.0, 0.0, 0.0};

        // shift forces
        double3 shiftForces_i = {0.0, 0.0, 0.0};

        // get atom type
        size_t atomType = atomTypes[i];
        size_t vDWType  = internalGlobalVDWTypes[i];

        // get positions of i-th molecule
        double3 position_i = {
            positions[3 * i],
            positions[3 * i + 1],
            positions[3 * i + 2]
        };

        for (size_t j = 0; j < numberOfMolecules; ++j)
        {
            double3 position_j = {
                positions[3 * j],
                positions[3 * j + 1],
                positions[3 * j + 2]
            };
        }
    }
}

/**
 * @brief calculate shift vector
 *
 * @param dxyz
 * @param box
 * @return double3
 */
__device__ double3 calculateShiftVector(double3 dxyz, double3 boxDimensions)
{
    double3 txyz;
    txyz.x = -boxDimensions.x * round(dxyz.x / boxDimensions.x);
    txyz.y = -boxDimensions.y * round(dxyz.y / boxDimensions.y);
    txyz.z = -boxDimensions.z * round(dxyz.z / boxDimensions.z);

    return txyz;
}

/**
 * @brief calculates forces, coulombic and non-coulombic energy for CUDA
 * routine.
 *
 * @param simBox
 * @param physicalData
 */
inline void PotentialCuda::
    calculateForces(simulationBox::SimulationBox &simBox, physicalData::PhysicalData &physicalData, simulationBox::CellList &)
{
    // start transfer timings -------------------------------------------------
    startTimingsSection("InterNonBonded - Transfer");

    // set total coulombic and non-coulombic energy
    double totalCoulombEnergy    = 0.0;
    double totalNonCoulombEnergy = 0.0;

    // get box parameters
    const auto h_boxDimensions = simBox.getBox().getBoxDimensions();
    double3    d_boxDimensions = {
        h_boxDimensions[0],
        h_boxDimensions[1],
        h_boxDimensions[2]
    };

    // get simulation parameters from simulation box
    const auto h_atomTypes       = simBox.flattenAtomTypes();
    const auto h_moleculeIndices = simBox.getMoleculeIndices();
    const auto h_internalGlobalVDWTypes =
        simBox.flattenInternalGlobalVDWTypes();
    const auto h_molTypes       = simBox.flattenMolTypes();
    const auto h_partialCharges = simBox.flattenPartialCharges();
    const auto h_positions      = simBox.flattenPositions();

    // TODO: check if forces are set to zero
    const auto h_forces = simBox.flattenForces();

    // initialize device memory
    size_t *d_atomTypes;
    size_t *d_moleculeIndices;
    size_t *d_internalGlobalVDWTypes;
    size_t *d_molTypes;
    double *d_partialCharges;
    double *d_positions;
    double *d_forces;

    // allocate memory on device
    hipMallocManaged(&d_atomTypes, h_atomTypes.size() * sizeof(size_t));
    hipMallocManaged(
        &d_moleculeIndices,
        h_moleculeIndices.size() * sizeof(size_t)
    );
    hipMallocManaged(
        &d_internalGlobalVDWTypes,
        h_internalGlobalVDWTypes.size() * sizeof(size_t)
    );
    hipMallocManaged(
        &d_internalGlobalVDWTypes,
        h_internalGlobalVDWTypes.size() * sizeof(size_t)
    );
    hipMallocManaged(&d_molTypes, h_molTypes.size() * sizeof(size_t));
    hipMallocManaged(
        &d_partialCharges,
        h_partialCharges.size() * sizeof(double)
    );
    hipMallocManaged(&d_positions, h_positions.size() * sizeof(double));
    hipMallocManaged(&d_forces, h_forces.size() * sizeof(double));

    // get number of atoms
    const size_t numberOfMolecules = simBox.getNumberOfMolecules();

    // end transfer timings ---------------------------------------------------
    stopTimingsSection("InterNonBonded - Transfer");

    // start calculation timings ----------------------------------------------
    startTimingsSection("InterNonBonded");

    size_t block_size = 256;
    size_t grid_size  = (numberOfMolecules + block_size - 1) / block_size;

    // calculate forces on device
    calculateForcesKernel<<<grid_size, block_size>>>(
        d_boxDimensions,
        d_atomTypes,
        d_moleculeIndices,
        d_internalGlobalVDWTypes,
        d_molTypes,
        d_partialCharges,
        d_positions,
        d_forces,
        numberOfMolecules,
        &totalCoulombEnergy,
        &totalNonCoulombEnergy
    );

    // synchronize device
    hipDeviceSynchronize();

    // stop calculation timings ------------------------------------------------
    stopTimingsSection("InterNonBonded");

    // start transfer timings --------------------------------------------------
    startTimingsSection("InterNonBonded - Transfer");

    // half energy due to double counting
    totalCoulombEnergy    *= 0.5;
    totalNonCoulombEnergy *= 0.5;

    simBox.deFlattenForces(h_forces);
    // TODO: check if shift forces transfer is needed

    // set total coulombic and non-coulombic energy
    physicalData.setCoulombEnergy(totalCoulombEnergy);
    physicalData.setNonCoulombEnergy(totalNonCoulombEnergy);

    // stop transfer timings ---------------------------------------------------
    stopTimingsSection("InterNonBonded - Transfer");

    // free memory on device
    hipFree(d_atomTypes);
    hipFree(d_moleculeIndices);
    hipFree(d_internalGlobalVDWTypes);
    hipFree(d_molTypes);
    hipFree(d_partialCharges);
    hipFree(d_positions);
    hipFree(d_forces);

    return;
}
